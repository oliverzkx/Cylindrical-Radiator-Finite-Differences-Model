#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


// CUDA kernel for 1D heat propagation (row-wise only)
__global__ void heat_kernel(float* next, const float* prev, int n, int m) {
    // Compute global coordinates for each thread
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int idx = i * m + j;

    // Prevent out-of-bounds memory access
    if (i >= n || j >= m || j < 2 || j >= m - 2) return;


    if (i < 2 && j < 6) {
        printf("GPU debug i=%d j=%d idx=%d prev[idx - 2]=%.4f\n", i, j, idx, prev[idx - 2]);
    }


    // Update only non-border columns
    if (j >= 2 && j < m - 2) {
        next[idx] = (
            1.60f * prev[idx - 2] +
            1.55f * prev[idx - 1] +
            1.00f * prev[idx]     +
            0.60f * prev[idx + 1] +
            0.25f * prev[idx + 2]
        ) / 5.0f;
    } else {
        // Preserve fixed boundary values
        next[idx] = prev[idx];
    }
}



// CUDA kernel to compute the average temperature of each row
__global__ void row_avg_kernel(float* data, float* row_avg, int n, int m) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;

    float sum = 0.0f;
    for (int j = 0; j < m; ++j)
        sum += data[row * m + j];

    row_avg[row] = sum / m;
}

// Host function to perform heat propagation using GPU
extern "C" void launch_cuda_heat(float* host_prev, int n, int m, int p, bool use_stop, float stop_avg, bool show_timing) {
    // Allocate device pointers
    float *d_prev, *d_next, *d_avg;

    // CUDA events for timing
    hipEvent_t start_total, stop_total;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_avg_kernel, stop_avg_kernel;
    hipEvent_t start_h2d, stop_h2d, start_d2h, stop_d2h;

    // Create all timing events
    hipEventCreate(&start_total);     hipEventCreate(&stop_total);
    hipEventCreate(&start_kernel);    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_avg_kernel);hipEventCreate(&stop_avg_kernel);
    hipEventCreate(&start_h2d);       hipEventCreate(&stop_h2d);
    hipEventCreate(&start_d2h);       hipEventCreate(&stop_d2h);

    // Start total execution timer
    hipEventRecord(start_total);

    // Allocate memory on GPU
    hipMalloc(&d_prev, n * m * sizeof(float));
    hipMalloc(&d_next, n * m * sizeof(float));
    hipMalloc(&d_avg, n * sizeof(float));

    // Copy initial data to GPU and time it
    hipEventRecord(start_h2d);
    hipMemcpy(d_prev, host_prev, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop_h2d);
    hipEventSynchronize(stop_h2d);

    // Kernel launch configuration
    dim3 block(16, 16);
    dim3 grid((m + block.x - 1) / block.x, (n + block.y - 1) / block.y);

    // Allocate host-side buffer for row averages
    std::vector<float> h_avg(n);

    float total_avg_time = 0.0f;

    // Start kernel timing
    hipEventRecord(start_kernel);

    for (int step = 0; step < p; ++step) {
        heat_kernel<<<grid, block>>>(d_next, d_prev, n, m);
        //  Add this immediately after kernel launch
        hipDeviceSynchronize();
        std::swap(d_prev, d_next);

        if (use_stop) {
            // Time row average calculation
            hipEventRecord(start_avg_kernel);
            row_avg_kernel<<<(n + 255) / 256, 256>>>(d_prev, d_avg, n, m);
            hipEventRecord(stop_avg_kernel);
            hipEventSynchronize(stop_avg_kernel);

            float avg_time = 0.0f;
            hipEventElapsedTime(&avg_time, start_avg_kernel, stop_avg_kernel);
            total_avg_time += avg_time;

            // Copy row averages back and check stopping condition
            hipMemcpy(h_avg.data(), d_avg, n * sizeof(float), hipMemcpyDeviceToHost);
            for (int i = 0; i < n; ++i) {
                if (h_avg[i] >= stop_avg) {
                    std::cout << "🔴 Stopped at iteration " << step + 1
                              << " due to average temp >= " << stop_avg << "\n";
                    goto END_KERNEL;
                }
            }
        }
    }

END_KERNEL:
    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);

    // Copy final matrix back to host
    hipEventRecord(start_d2h);
    hipMemcpy(host_prev, d_prev, n * m * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop_d2h);
    hipEventSynchronize(stop_d2h);

    // Stop total timer
    hipEventRecord(stop_total);
    hipEventSynchronize(stop_total);

    // Compute elapsed times
    float t_total, t_kernel, t_avg, t_h2d, t_d2h;
    hipEventElapsedTime(&t_total, start_total, stop_total);
    hipEventElapsedTime(&t_kernel, start_kernel, stop_kernel);
    hipEventElapsedTime(&t_h2d, start_h2d, stop_h2d);
    hipEventElapsedTime(&t_d2h, start_d2h, stop_d2h);
    t_avg = total_avg_time;

    if (show_timing) {
        std::cout << "[GPU] Memcpy H2D:       " << t_h2d    << " ms\n";
        std::cout << "[GPU] Propagation Time: " << t_kernel << " ms\n";
        std::cout << "[GPU] Row Average Time: " << t_avg    << " ms\n";
        std::cout << "[GPU] Memcpy D2H:       " << t_d2h    << " ms\n";
        std::cout << "[GPU] Total Time:       " << t_total  << " ms\n";
    }

    // Cleanup
    hipFree(d_prev); hipFree(d_next); hipFree(d_avg);
    hipEventDestroy(start_total);     hipEventDestroy(stop_total);
    hipEventDestroy(start_kernel);    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_avg_kernel);hipEventDestroy(stop_avg_kernel);
    hipEventDestroy(start_h2d);       hipEventDestroy(stop_h2d);
    hipEventDestroy(start_d2h);       hipEventDestroy(stop_d2h);
}


