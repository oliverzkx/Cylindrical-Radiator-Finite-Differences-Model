#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


// CUDA kernel for 1D heat propagation (row-wise only)
__global__ void heat_kernel(float* prev, float* next, int n, int m) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n || col < 2 || col >= m - 2) return;

    int idx = row * m + col;
    next[idx] = (1.60f * prev[idx - 2] +
                 1.55f * prev[idx - 1] +
                 0.60f * prev[idx + 1] +
                 0.25f * prev[idx + 2]) / 5.0f;
}

// CUDA kernel to compute the average temperature of each row
__global__ void row_avg_kernel(float* data, float* row_avg, int n, int m) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;

    float sum = 0.0f;
    for (int j = 0; j < m; ++j)
        sum += data[row * m + j];

    row_avg[row] = sum / m;
}

// Host function to perform heat propagation using GPU
extern "C" void launch_cuda_heat(float* host_prev, int n, int m, int p, bool use_stop, float stop_avg, bool show_timing) {
    float *d_prev, *d_next, *d_avg;
    hipEvent_t start_total, stop_total;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_avg_kernel, stop_avg_kernel;
    hipEvent_t start_h2d, stop_h2d, start_d2h, stop_d2h;

    // Create events for timing
    hipEventCreate(&start_total);  hipEventCreate(&stop_total);
    hipEventCreate(&start_kernel); hipEventCreate(&stop_kernel);
    hipEventCreate(&start_avg_kernel); hipEventCreate(&stop_avg_kernel);
    hipEventCreate(&start_h2d);    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_d2h);    hipEventCreate(&stop_d2h);

    // Start total timer
    hipEventRecord(start_total);

    // Allocate memory
    hipMalloc(&d_prev, n * m * sizeof(float));
    hipMalloc(&d_next, n * m * sizeof(float));
    hipMalloc(&d_avg, n * sizeof(float));

    // H2D timing
    hipEventRecord(start_h2d);
    hipMemcpy(d_prev, host_prev, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop_h2d);
    hipEventSynchronize(stop_h2d);

    // Grid setup
    dim3 block(16, 16);
    dim3 grid((m + 15) / 16, (n + 15) / 16);
    std::vector<float> h_avg(n);

    float total_avg_time = 0.0f;

    // Start kernel timing
    hipEventRecord(start_kernel);

    for (int step = 0; step < p; ++step) {
        heat_kernel<<<grid, block>>>(d_prev, d_next, n, m);
        std::swap(d_prev, d_next);

        if (use_stop) {
            // Independent timing for row average kernel
            hipEventRecord(start_avg_kernel);
            row_avg_kernel<<<(n + 255) / 256, 256>>>(d_prev, d_avg, n, m);
            hipEventRecord(stop_avg_kernel);
            hipEventSynchronize(stop_avg_kernel);

            float avg_time = 0.0f;
            hipEventElapsedTime(&avg_time, start_avg_kernel, stop_avg_kernel);
            total_avg_time += avg_time;

            hipMemcpy(h_avg.data(), d_avg, n * sizeof(float), hipMemcpyDeviceToHost);
            for (int i = 0; i < n; ++i) {
                if (h_avg[i] >= stop_avg) {
                    std::cout << "🛑 Stopped at iteration " << step + 1
                              << " due to average temp >= " << stop_avg << "\n";
                    goto END_KERNEL;
                }
            }
        }
    }

END_KERNEL:
    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);

    // D2H timing
    hipEventRecord(start_d2h);
    hipMemcpy(host_prev, d_prev, n * m * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop_d2h);
    hipEventSynchronize(stop_d2h);

    // Stop total timer
    hipEventRecord(stop_total);
    hipEventSynchronize(stop_total);

    // Read elapsed times
    float t_total, t_kernel, t_avg, t_h2d, t_d2h;
    hipEventElapsedTime(&t_total, start_total, stop_total);
    hipEventElapsedTime(&t_kernel, start_kernel, stop_kernel);
    hipEventElapsedTime(&t_h2d, start_h2d, stop_h2d);
    hipEventElapsedTime(&t_d2h, start_d2h, stop_d2h);
    t_avg = total_avg_time;

    // Output timing results
    std::cout << "[GPU] Memcpy H2D:       " << t_h2d    << " ms\n";
    std::cout << "[GPU] Propagation Time: " << t_kernel << " ms\n";
    std::cout << "[GPU] Row Average Time: " << t_avg    << " ms\n";
    std::cout << "[GPU] Memcpy D2H:       " << t_d2h    << " ms\n";
    std::cout << "[GPU] Total Time:       " << t_total  << " ms\n";

    // Cleanup
    hipFree(d_prev); hipFree(d_next); hipFree(d_avg);
    hipEventDestroy(start_total);  hipEventDestroy(stop_total);
    hipEventDestroy(start_kernel); hipEventDestroy(stop_kernel);
    hipEventDestroy(start_avg_kernel); hipEventDestroy(stop_avg_kernel);
    hipEventDestroy(start_h2d);    hipEventDestroy(stop_h2d);
    hipEventDestroy(start_d2h);    hipEventDestroy(stop_d2h);
}

