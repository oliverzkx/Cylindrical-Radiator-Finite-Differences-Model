#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// CUDA kernel for 1D heat propagation (row-wise only)
__global__ void heat_kernel(float* prev, float* next, int n, int m) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= n || col < 2 || col >= m - 2) return;

    int idx = row * m + col;
    next[idx] = (1.60f * prev[idx - 2] +
                 1.55f * prev[idx - 1] +
                 0.60f * prev[idx + 1] +
                 0.25f * prev[idx + 2]) / 5.0f;
}

// CUDA kernel to compute the average temperature of each row
__global__ void row_avg_kernel(float* data, float* row_avg, int n, int m) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n) return;

    float sum = 0.0f;
    for (int j = 0; j < m; ++j)
        sum += data[row * m + j];

    row_avg[row] = sum / m;
}

// Host function to perform heat propagation using GPU
void launch_cuda_heat(float* host_prev, int n, int m, int p, bool use_stop, float stop_avg) {
    float *d_prev, *d_next, *d_avg;
    hipEvent_t start_total, stop_total, start_kernel, stop_kernel, start_h2d, stop_h2d, start_d2h, stop_d2h;

    // Create events
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);
    hipEventCreate(&start_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_h2d);
    hipEventCreate(&stop_h2d);
    hipEventCreate(&start_d2h);
    hipEventCreate(&stop_d2h);

    // Start total timer
    hipEventRecord(start_total);

    // Allocate device memory
    hipMalloc(&d_prev, n * m * sizeof(float));
    hipMalloc(&d_next, n * m * sizeof(float));
    hipMalloc(&d_avg, n * sizeof(float));

    // H2D timing
    hipEventRecord(start_h2d);
    hipMemcpy(d_prev, host_prev, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop_h2d);
    hipEventSynchronize(stop_h2d);

    // Configure CUDA execution
    dim3 block(16, 16);
    dim3 grid((m + 15) / 16, (n + 15) / 16);
    std::vector<float> h_avg(n);

    // Start kernel timing
    hipEventRecord(start_kernel);

    for (int step = 0; step < p; ++step) {
        heat_kernel<<<grid, block>>>(d_prev, d_next, n, m);
        std::swap(d_prev, d_next);

        if (use_stop) {
            row_avg_kernel<<<(n + 255) / 256, 256>>>(d_prev, d_avg, n, m);
            hipMemcpy(h_avg.data(), d_avg, n * sizeof(float), hipMemcpyDeviceToHost);
            for (int i = 0; i < n; ++i) {
                if (h_avg[i] >= stop_avg) {
                    std::cout << "🛑 Stopped at iteration " << step + 1
                              << " due to average temp >= " << stop_avg << "\n";
                    goto END_KERNEL;
                }
            }
        }
    }

END_KERNEL:
    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);

    // D2H timing
    hipEventRecord(start_d2h);
    hipMemcpy(host_prev, d_prev, n * m * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop_d2h);
    hipEventSynchronize(stop_d2h);

    // Stop total timer
    hipEventRecord(stop_total);
    hipEventSynchronize(stop_total);

    // Print timing results
    float t_h2d, t_kernel, t_d2h, t_total;
    hipEventElapsedTime(&t_h2d, start_h2d, stop_h2d);
    hipEventElapsedTime(&t_kernel, start_kernel, stop_kernel);
    hipEventElapsedTime(&t_d2h, start_d2h, stop_d2h);
    hipEventElapsedTime(&t_total, start_total, stop_total);

    std::cout << "[GPU] Memcpy H2D:       " << t_h2d    << " ms\n";
    std::cout << "[GPU] Propagation Time: " << t_kernel << " ms\n";
    std::cout << "[GPU] Memcpy D2H:       " << t_d2h    << " ms\n";
    std::cout << "[GPU] Total Time:       " << t_total  << " ms\n";

    // Cleanup
    hipFree(d_prev); hipFree(d_next); hipFree(d_avg);
    hipEventDestroy(start_total); hipEventDestroy(stop_total);
    hipEventDestroy(start_kernel); hipEventDestroy(stop_kernel);
    hipEventDestroy(start_h2d); hipEventDestroy(stop_h2d);
    hipEventDestroy(start_d2h); hipEventDestroy(stop_d2h);
}