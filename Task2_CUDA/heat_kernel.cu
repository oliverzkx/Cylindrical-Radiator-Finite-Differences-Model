#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


// CUDA kernel for 1D heat propagation (row-wise only)
// CUDA kernel for 2D heat propagation with 5-point stencil (row-wise only)
__global__ void heat_kernel(float* next, const float* prev, int n, int m) {
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= n || j >= m) return;

    int idx = i * m + j;

    // First: copy all values by default (makes it equivalent to CPU)
    next[idx] = prev[idx];

    // Then: only stencil region will overwrite with new value
    if (i >= 1 && i < n - 1 && j >= 2 && j < m - 2) {
        next[idx] =
            (1.60f * prev[i * m + (j - 2)] +
             1.55f * prev[i * m + (j - 1)] +
             1.00f * prev[i * m + j]     +
             0.60f * prev[i * m + (j + 1)] +
             0.25f * prev[i * m + (j + 2)]) / 5.0f;
    }
}




// CUDA kernel to compute the average temperature of each row
__global__ void row_avg_kernel(const float* data, float* row_avg, int n, int m) {
    int i = blockIdx.x;         // Each block handles one row
    int j = threadIdx.x;        // Each thread processes one column in the row

    if (i >= n || j >= m) return;

    // Shared memory to accumulate sum per row
    __shared__ float local_sum[1];

    // Initialize shared memory (only thread 0)
    if (j == 0) local_sum[0] = 0.0f;
    __syncthreads();

    // Each thread adds its column's value to shared row sum
    atomicAdd(&local_sum[0], data[i * m + j]);
    __syncthreads();

    // First thread in the block writes the average
    if (j == 0) {
        row_avg[i] = local_sum[0] / m;
    }
}



// Host function to perform heat propagation using GPU
extern "C" void launch_cuda_heat(float* host_prev, int n, int m, int p, bool use_stop, float stop_avg, bool show_timing) {
    // Allocate device pointers
    float *d_prev, *d_next, *d_avg;

    // CUDA events for timing
    hipEvent_t start_total, stop_total;
    hipEvent_t start_kernel, stop_kernel;
    hipEvent_t start_avg_kernel, stop_avg_kernel;
    hipEvent_t start_h2d, stop_h2d, start_d2h, stop_d2h;

    // Create all timing events
    hipEventCreate(&start_total);     hipEventCreate(&stop_total);
    hipEventCreate(&start_kernel);    hipEventCreate(&stop_kernel);
    hipEventCreate(&start_avg_kernel);hipEventCreate(&stop_avg_kernel);
    hipEventCreate(&start_h2d);       hipEventCreate(&stop_h2d);
    hipEventCreate(&start_d2h);       hipEventCreate(&stop_d2h);

    // Start total execution timer
    hipEventRecord(start_total);

    // Allocate memory on GPU
    hipMalloc(&d_prev, n * m * sizeof(float));
    hipMalloc(&d_next, n * m * sizeof(float));
    hipMalloc(&d_avg, n * sizeof(float));

    // Copy initial data to GPU and time it
    hipEventRecord(start_h2d);
    hipMemcpy(d_prev, host_prev, n * m * sizeof(float), hipMemcpyHostToDevice);
    hipEventRecord(stop_h2d);
    hipEventSynchronize(stop_h2d);

    // Kernel launch configuration
    dim3 block(16, 16);
    dim3 grid((m + block.x - 1) / block.x, (n + block.y - 1) / block.y);

    // Allocate host-side buffer for row averages
    std::vector<float> h_avg(n);

    float total_avg_time = 0.0f;

    // Start kernel timing
    hipEventRecord(start_kernel);

    for (int step = 0; step < p; ++step) {
        heat_kernel<<<grid, block>>>(d_next, d_prev, n, m);
        //  Add this immediately after kernel launch
        hipDeviceSynchronize();
        std::swap(d_prev, d_next);

        if (use_stop) {
            // Time row average calculation
            hipEventRecord(start_avg_kernel);
            row_avg_kernel<<<n, m>>>(d_prev, d_avg, n, m);
            hipEventRecord(stop_avg_kernel);
            hipEventSynchronize(stop_avg_kernel);

            float avg_time = 0.0f;
            hipEventElapsedTime(&avg_time, start_avg_kernel, stop_avg_kernel);
            total_avg_time += avg_time;

            // Copy row averages back and check stopping condition
            hipMemcpy(h_avg.data(), d_avg, n * sizeof(float), hipMemcpyDeviceToHost);
            for (int i = 0; i < n; ++i) {
                if (h_avg[i] >= stop_avg) {
                    std::cout << "🔴 Stopped at iteration " << step + 1
                              << " due to average temp >= " << stop_avg << "\n";
                    goto END_KERNEL;
                }
            }
        }
    }

END_KERNEL:
    hipEventRecord(stop_kernel);
    hipEventSynchronize(stop_kernel);

    // Copy final matrix back to host
    hipEventRecord(start_d2h);
    hipMemcpy(host_prev, d_prev, n * m * sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop_d2h);
    hipEventSynchronize(stop_d2h);

    // Stop total timer
    hipEventRecord(stop_total);
    hipEventSynchronize(stop_total);

    // Compute elapsed times
    float t_total, t_kernel, t_avg, t_h2d, t_d2h;
    hipEventElapsedTime(&t_total, start_total, stop_total);
    hipEventElapsedTime(&t_kernel, start_kernel, stop_kernel);
    hipEventElapsedTime(&t_h2d, start_h2d, stop_h2d);
    hipEventElapsedTime(&t_d2h, start_d2h, stop_d2h);
    t_avg = total_avg_time;

    if (show_timing) {
        std::cout << "[GPU] Memcpy H2D:       " << t_h2d    << " ms\n";
        std::cout << "[GPU] Propagation Time: " << t_kernel << " ms\n";
        std::cout << "[GPU] Row Average Time: " << t_avg    << " ms\n";
        std::cout << "[GPU] Memcpy D2H:       " << t_d2h    << " ms\n";
        std::cout << "[GPU] Total Time:       " << t_total  << " ms\n";
    }

    // Cleanup
    hipFree(d_prev); hipFree(d_next); hipFree(d_avg);
    hipEventDestroy(start_total);     hipEventDestroy(stop_total);
    hipEventDestroy(start_kernel);    hipEventDestroy(stop_kernel);
    hipEventDestroy(start_avg_kernel);hipEventDestroy(stop_avg_kernel);
    hipEventDestroy(start_h2d);       hipEventDestroy(stop_h2d);
    hipEventDestroy(start_d2h);       hipEventDestroy(stop_d2h);
}


